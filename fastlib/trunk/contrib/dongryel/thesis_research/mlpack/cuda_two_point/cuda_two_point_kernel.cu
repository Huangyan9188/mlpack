/** @file cuda_two_point_kernel.cu
 *
 *  CUDA implementation of two point.
 *
 *  @author Bill March (march@gatech.edu)
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ float PointDistanceSqr(float* query_point, float* reference_point) {

  float x_dist = query_point[0] - reference_point[0];
  float y_dist = query_point[1] - reference_point[1];
  float z_dist = query_point[2] - reference_point[2];
  
  return (x_dist * x_dist + y_dist * y_dist + z_dist * z_dist);

} // PointDistanceSqr()

__device__ void LoadReferencePoint(
  float *reference,
  int reference_point_id,
  float *reference_point_shared_mem) {

  int num_dimensions = 3;

  int i;
  int source_pos = reference_point_id * num_dimensions;
  int dest_pos = threadIdx.x * num_dimensions;
  for(i = 0; i < num_dimensions; i++, dest_pos++, source_pos++) {
    reference_point_shared_mem[dest_pos] = reference[source_pos];
  }

// why are we resetting these?
  source_pos = reference_point_id * num_dimensions;
  dest_pos = threadIdx.x * num_dimensions;
} // LoadReferencePoint

__device__ void TestPointPair(
  float *query_point,
  float *reference_point,
  int *local_sum, float lower_bound_sqr, float upper_bound_sqr) {

  float squared_distance = PointDistanceSqr(query_point, reference_point);
  
  *local_sum += (lower_bound_sqr <= squared_distance) 
                && (squared_distance <= upper_bound_sqr);

} // TestPointPair

__device__ void TileBaseCase(
  float *query_point,
  int num_reference_points_in_this_tile,
  float *reference_point_shared_mem,
  int *local_sum, float lower_bound_sqr, float upper_bound_sqr) {

  int num_dimensions = 3;

  int i;
  float *reference_point = reference_point_shared_mem;
  for(i = 0; i < num_reference_points_in_this_tile; i++,
      reference_point += num_dimensions) {
    
    TestPointPair(query_point, reference_point, local_sum,
                  lower_bound_sqr, upper_bound_sqr);
  
  } // for i
} // TileBaseCase


__global__ void TwoPointKernelOnDevice(
  float *query, int num_query_points,
  float *reference, int num_reference_points,
  int *two_point_sums_out,
  float lower_bound_sqr, float upper_bound_sqr) {

  int num_dimensions = 3;

  // The shared memory that is used to load a list of reference
  // points.
  float query_point_local_mem[3];
  __shared__ float reference_point_shared_mem[1024];

  // Load the assigned query point to the local memory.
  int global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = num_dimensions * global_thread_id;
  int i, tile;
  if(global_thread_id < num_query_points) {
    for(i = 0; i < num_dimensions; i++, offset++) {
      query_point_local_mem[i] = query[offset];
    }
  }

  // Local variable for accumulating the kernel sum.
  int local_sum = 0;

  // The number of reference points in each round is the minimum of
  // the two quantities: the number of points that can be packed in
  // the shared memory and the number of threads available.
  int num_reference_points_per_tile =
    min(1024 / num_dimensions, blockDim.x);

  for(i = 0, tile = 0; i < num_reference_points;
      i += num_reference_points_per_tile, tile++) {

    // Each thread loads the specified number of points, and
    // synchronize all threads within this block before computing.
    int reference_point_id = tile * num_reference_points_per_tile + threadIdx.x;
    int num_reference_points_in_this_tile =
      min(num_reference_points - i, num_reference_points_per_tile);
    int ending_rpoint_id = tile * num_reference_points_per_tile +
                           num_reference_points_in_this_tile;

    if(reference_point_id < ending_rpoint_id) {
      LoadReferencePoint(reference, reference_point_id,
                         reference_point_shared_mem);
    }
    __syncthreads();

    // Accumulate the kernel sum contribution of the current tile.
    if(global_thread_id < num_query_points) {
      TileBaseCase(query_point_local_mem,
                   num_reference_points_in_this_tile,
                   reference_point_shared_mem, &local_sum, lower_bound_sqr, 
                   upper_bound_sqr);
    }

    // Synchronize all threads within this block before loading new
    // sets of points.
    __syncthreads();
  }

  if(global_thread_id < num_query_points) {
    two_point_sums_out[ global_thread_id ] = local_sum;
  }
}

extern "C" {

  void TwoPointKernelOnHost(
    double *query, int num_query_points,
    double *reference, int num_reference_points,
    int *two_point_sums_out, float lower_bound_sqr, float upper_bound_sqr) {

    int num_dimensions = 3;

    // Prepare to copy the points into single precision format on the
    // GPU.
    float *query_on_host = new float[
      num_dimensions * num_query_points ];
    float *query_on_device = NULL;
    float *reference_on_host = new float[
      num_dimensions * num_reference_points ];
    float *reference_on_device = NULL;
    int *two_point_sums_out_on_device = NULL;
    int num_query_bytes = num_query_points * num_dimensions * sizeof(float);
    int num_reference_bytes =
      num_reference_points * num_dimensions * sizeof(float);
    if(hipSuccess != hipMalloc(&query_on_device, num_query_bytes)) {
      printf("Error in allocating the query on the GPU.\n");
      return;
    }
    if(hipSuccess != hipMalloc(&reference_on_device, num_reference_bytes)) {
      printf("Error in allocating the reference on the GPU.\n");
      return;
    }
    if(hipSuccess !=
        hipMalloc(
          &two_point_sums_out_on_device, num_query_points * sizeof(float))) {
      printf("Error in allocating the kernel sum slots on the GPU.\n");
      return;
    }
    int i, j;
    int pos = 0;
    for(i = 0; i < num_query_points; i++) {
      for(j = 0; j < num_dimensions; j++, pos++) {
        query_on_host[pos] = query[pos];
      }
    }
    hipMemcpy(
      query_on_device, query_on_host, num_query_bytes, hipMemcpyHostToDevice);
    pos = 0;
    for(i = 0; i < num_reference_points; i++) {
      for(j = 0; j < num_dimensions; j++, pos++) {
        reference_on_host[pos] = reference[pos];
      }
    }
    hipMemcpy(
      reference_on_device, reference_on_host,
      num_reference_bytes, hipMemcpyHostToDevice);

    int num_threads_per_block = 512;
    int num_blocks = (num_query_points + num_threads_per_block - 1) /
                     num_threads_per_block;

    // Call the CUDA kernel.
    TwoPointKernelOnDevice <<< num_blocks, num_threads_per_block >>>(
      query_on_device, num_query_points,
      reference_on_device, num_reference_points, two_point_sums_out_on_device,
      lower_bound_sqr, upper_bound_sqr);

    // Copy out the result from the device to the host.
    hipMemcpy(
      two_point_sums_out, two_point_sums_out_on_device,
      num_query_points * sizeof(float), hipMemcpyDeviceToHost);

    // Free memory.
    delete[] query_on_host;
    delete[] reference_on_host;
  }
}
