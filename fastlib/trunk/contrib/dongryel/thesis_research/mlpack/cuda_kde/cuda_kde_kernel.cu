/** @file cuda_kde_kernel.cu
 *
 *  CUDA implementation of KDE.
 *
 *  @author Dongryeol Lee (dongryel@cc.gatech.edu)
 */

#include <hip/hip_runtime.h>

__device__ void LoadReferencePoint_(
  int num_dimensions,
  float *reference,
  int reference_point_id) {

  extern __shared__ float reference_point_shared_mem[];

  int i;
  int source_pos = reference_point_id * num_dimensions;
  int dest_pos = threadIdx.x * num_dimensions;
  for(i = 0; i < num_dimensions; i++, dest_pos++, source_pos++) {
    reference_point_shared_mem[dest_pos] = reference[source_pos];
  }
}

__device__ void AccumulateReferencePointContribution_(
  int num_dimensions,
  float bandwidth,
  float *query_point,
  float *reference_point,
  float *local_sum) {

  int i;
  float squared_distance = 0.0;
  for(i = 0; i < num_dimensions; i++) {
    float diff = (query_point[i] - reference_point[i]) / bandwidth;
    squared_distance += diff * diff;
  }
  float kernel_value = expf(- squared_distance * 0.5);
  (*local_sum) += kernel_value;
}

__device__ void AccumulateTileContribution_(
  int num_dimensions,
  float bandwidth,
  float *query_point,
  float *local_sum) {

  extern __shared__ float reference_point_shared_mem[];

  int i;
  float *reference_point = reference_point_shared_mem;
  for(i = 0; i < blockDim.x; i++, reference_point += num_dimensions) {
    AccumulateReferencePointContribution_(
      num_dimensions, bandwidth, query_point, reference_point, local_sum);
  }
}

__global__ void NbodyKernelOnDevice(
  int num_dimensions,
  float bandwidth,
  float *query, int num_query_points,
  float *reference, int num_reference_points,
  float *kernel_sums_out) {

  // The shared memory that is used to load a list of reference
  // points.
  float query_point_local_mem[100];
  __shared__ float reference_point_shared_mem[1024];

  // Load the assigned query point to the local memory.
  int global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = num_dimensions * global_thread_id;
  int i, tile;
  for(i = 0; i < num_dimensions; i++, offset++) {
    query_point_local_mem[i] = query[offset];
  }

  // Local variable for accumulating the kernel sum.
  float local_sum = 0.0;

  for(i = 0, tile = 0; i < num_reference_points; i += blockDim.x, tile++) {

    // Each thread loads the specified number of points, and
    // synchronize all threads within this block before computing.
    int reference_point_id = tile * blockDim.x + threadIdx.x;
    LoadReferencePoint_(num_dimensions, reference, reference_point_id);
    __syncthreads();

    // Accumulate the kernel sum contribution of the current tile.
    AccumulateTileContribution_(
      num_dimensions, bandwidth, query_point_local_mem, & local_sum);

    // Synchronize all threads within this block before loading new
    // sets of points.
    __syncthreads();
  }

  kernel_sums_out[ global_thread_id ] = local_sum;
}

extern "C" {

  void NbodyKernelOnHost(
    int num_dimensions,
    float bandwidth,
    float *query, int num_query_points,
    float *reference, int num_reference_points,
    float *kernel_sums_out) {

    // Query the number of multiprocessors on the GPU.
    hipDeviceProp_t prop;
    int count;
    hipGetDeviceCount(& count);
    hipGetDeviceProperties(&prop, 0);

    int num_blocks = prop.multiProcessorCount;
    int num_threads_per_block = num_dimensions / num_blocks;
    NbodyKernelOnDevice <<< num_blocks, num_threads_per_block >>>(
      num_dimensions, bandwidth, query, num_query_points,
      reference, num_reference_points, kernel_sums_out);
  }
}
