/** @file cuda_kde_kernel.cu
 *
 *  CUDA implementation of KDE.
 *
 *  @author Dongryeol Lee (dongryel@cc.gatech.edu)
 */

#include <hip/hip_runtime.h>

__global__ void NbodyKernelOnDevice(
  float *query, int num_query_points,
  float *reference, int num_reference_points) {

  // The shared memory that is used to load a list of reference
  // points.
  __shared__ float reference_point_shared_mem[1024];

  int i = 3;
}

extern "C" {

  void NbodyKernelOnHost(
    float *query, int num_query_points,
    float *reference, int num_reference_points) {
    NbodyKernelOnDevice <<< 1, 1>>>(
      query, num_query_points, reference, num_reference_points);
  }
}
