/** @file cuda_kde_kernel.cu
 *
 *  CUDA implementation of KDE.
 *
 *  @author Dongryeol Lee (dongryel@cc.gatech.edu)
 */

#include <hip/hip_runtime.h>

__global__ void NbodyKernelOnDevice(
  int num_dimensions,
  float *query, int num_query_points,
  float *reference, int num_reference_points) {

  // The shared memory that is used to load a list of reference
  // points.
  float query_point_local_mem[10];
  __shared__ float reference_point_shared_mem[1024];

  // Load the assigned query point to the local memory.
  int global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = num_dimensions * global_thread_id;
  int i;
  for(i = 0; i < num_dimensions; i++, offset++) {
    query_point_local_mem[i] = query[offset];
  }
}

extern "C" {

  void NbodyKernelOnHost(
    int num_dimensions,
    float *query, int num_query_points,
    float *reference, int num_reference_points) {

    // Query the number of multiprocessors on the GPU.
    hipDeviceProp_t prop;
    int count;
    hipGetDeviceCount(& count);
    hipGetDeviceProperties(&prop, 0);

    int num_blocks = prop.multiProcessorCount;
    int num_threads_per_block = num_dimensions / num_blocks;
    NbodyKernelOnDevice <<< num_blocks, num_threads_per_block >>>(
      num_dimensions, query, num_query_points, reference, num_reference_points);
  }
}
