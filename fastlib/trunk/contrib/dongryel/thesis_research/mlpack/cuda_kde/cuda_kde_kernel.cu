/** @file cuda_kde_kernel.cu
 *
 *  CUDA implementation of KDE.
 *
 *  @author Dongryeol Lee (dongryel@cc.gatech.edu)
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ void LoadReferencePoint_(
  int num_dimensions,
  float *reference,
  int reference_point_id) {

  extern __shared__ float reference_point_shared_mem[];

  int i;
  int source_pos = reference_point_id * num_dimensions;
  int dest_pos = threadIdx.x * num_dimensions;
  for(i = 0; i < num_dimensions; i++, dest_pos++, source_pos++) {
    reference_point_shared_mem[dest_pos] = reference[source_pos];
  }
}

__device__ void AccumulateReferencePointContribution_(
  int num_dimensions,
  float bandwidth,
  float *query_point,
  float *reference_point,
  float *local_sum) {

  int i;
  float squared_distance = 0.0;
  for(i = 0; i < num_dimensions; i++) {
    float diff = (query_point[i] - reference_point[i]) / bandwidth;
    squared_distance += diff * diff;
  }
  float kernel_value = expf(- squared_distance * 0.5);
  (*local_sum) += kernel_value;
}

__device__ void AccumulateTileContribution_(
  int num_dimensions,
  float bandwidth,
  float *query_point,
  int num_reference_points_in_this_tile,
  float *local_sum) {

  extern __shared__ float reference_point_shared_mem[];

  int i;
  float *reference_point = reference_point_shared_mem;
  for(i = 0; i < num_reference_points_in_this_tile; i++,
      reference_point += num_dimensions) {
    AccumulateReferencePointContribution_(
      num_dimensions, bandwidth, query_point, reference_point, local_sum);
  }
}

__global__ void NbodyKernelOnDevice(
  int num_dimensions,
  float bandwidth,
  float *query, int num_query_points,
  float *reference, int num_reference_points,
  float *kernel_sums_out) {

  // The shared memory that is used to load a list of reference
  // points.
  float query_point_local_mem[100];
  __shared__ float reference_point_shared_mem[1024];

  // Load the assigned query point to the local memory.
  int global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  int offset = num_dimensions * global_thread_id;
  int i, tile;
  if(global_thread_id < num_query_points) {
    for(i = 0; i < num_dimensions; i++, offset++) {
      query_point_local_mem[i] = query[offset];
    }
  }

  // Local variable for accumulating the kernel sum.
  float local_sum = 0.0;
  int num_reference_points_per_tile = 1024 / num_dimensions;

  for(i = 0, tile = 0; i < num_reference_points;
      i += num_reference_points_per_tile, tile++) {

    // Each thread loads the specified number of points, and
    // synchronize all threads within this block before computing.
    int reference_point_id = tile * num_reference_points_per_tile + threadIdx.x;
    int num_reference_points_in_this_tile =
      min(
        num_reference_points - i, num_reference_points_per_tile);
    int ending_rpoint_id = tile * num_reference_points_per_tile +
                           num_reference_points_in_this_tile;

    if(reference_point_id < ending_rpoint_id) {
      LoadReferencePoint_(num_dimensions, reference, reference_point_id);
    }
    __syncthreads();

    // Accumulate the kernel sum contribution of the current tile.
    if(global_thread_id < num_query_points) {
      AccumulateTileContribution_(
        num_dimensions, bandwidth, query_point_local_mem,
        num_reference_points_in_this_tile, & local_sum);
    }

    // Synchronize all threads within this block before loading new
    // sets of points.
    __syncthreads();
  }

  if(global_thread_id < num_query_points) {
    kernel_sums_out[ global_thread_id ] = local_sum;
  }
}

extern "C" {

  void NbodyKernelOnHost(
    int num_dimensions,
    float bandwidth,
    double *query, int num_query_points,
    double *reference, int num_reference_points,
    float *kernel_sums_out) {

    // Prepare to copy the points into single precision format on the
    // GPU.
    float *query_on_host = new float[
      num_dimensions * num_query_points ];
    float *query_on_device = NULL;
    float *reference_on_host = new float[
      num_dimensions * num_reference_points ];
    float *reference_on_device = NULL;
    float *kernel_sums_out_device = NULL;
    int num_query_bytes = num_query_points * num_dimensions * sizeof(float);
    int num_reference_bytes =
      num_reference_points * num_dimensions * sizeof(float);
    if(hipSuccess != hipMalloc(&query_on_device, num_query_bytes)) {
      printf("Error in allocating the query on the GPU.\n");
      return;
    }
    if(hipSuccess != hipMalloc(&reference_on_device, num_reference_bytes)) {
      printf("Error in allocating the reference on the GPU.\n");
      return;
    }
    if(hipSuccess !=
        hipMalloc(
          &kernel_sums_out_device, num_query_points * sizeof(float))) {
      printf("Error in allocating the kernel sum slots on the GPU.\n");
      return;
    }
    int i, j;
    int pos = 0;
    for(i = 0; i < num_query_points; i++) {
      for(j = 0; j < num_dimensions; j++, pos++) {
        query_on_host[pos] = query[pos];
      }
    }
    hipMemcpy(
      query_on_device, query_on_host, num_query_bytes, hipMemcpyHostToDevice);
    pos = 0;
    for(i = 0; i < num_reference_points; i++) {
      for(j = 0; j < num_dimensions; j++, pos++) {
        reference_on_host[pos] = reference[pos];
      }
    }
    hipMemcpy(
      reference_on_device, reference_on_host,
      num_reference_bytes, hipMemcpyHostToDevice);

    int num_threads_per_block = 512;
    int num_blocks = (num_query_points + num_threads_per_block - 1) /
                     num_threads_per_block;

    // Call the CUDA kernel.
    NbodyKernelOnDevice <<< num_blocks, num_threads_per_block >>>(
      num_dimensions, bandwidth, query_on_device, num_query_points,
      reference_on_device, num_reference_points, kernel_sums_out_device);

    // Copy out the result from the device to the host.
    hipMemcpy(
      kernel_sums_out, kernel_sums_out_device,
      num_query_points * sizeof(float), hipMemcpyDeviceToHost);

    // Free memory.
    delete[] query_on_host;
    delete[] reference_on_host;
  }
}
